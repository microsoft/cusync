#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

//<OPTIMIZATIONS>
//</OPTIMIZATIONS>

// #define LLAMA

#if defined(TILESYNC)
#if !defined(LLAMA)
  #define NO_ATOMIC_ADD
#endif
#define REORDER_TILE_LOADS
#endif

// #define AVOID_CUSTOM_ORDER
// #define AVOID_WAIT_KERNEL

// #if defined(TILESYNC) || defined(TILEBATCH)
// #define AVOID_CUSTOM_ORDER
// #define AVOID_WAIT_KERNEL
// #endif 

#include<cuSync.h>

#ifdef ROWSYNC
  using ProdCuStage = CuStage<CuStageType::Producer, RowMajor, RowSync>;
  using MiddleCuStage = CuStage<CuStageType::Producer | CuStageType::Consumer, RowMajor, RowSync>;
  using ConsCuStage = CuStage<CuStageType::Consumer, RowMajor, RowSync>;
  using Sync = RowSync;
#elif defined(TILEBATCH)
  using ProdCuStage = CuStage<CuStageType::Producer, RowMajor, TileSync<2>>;
  using MiddleCuStage = CuStage<CuStageType::Producer | CuStageType::Consumer, RowMajor, TileSync<2>>;
  using ConsCuStage = CuStage<CuStageType::Consumer, RowMajor, TileSync<2>>;
  using Sync = TileSync<2>;
#elif defined(TILESYNC)
  using ProdCuStage = CuStage<CuStageType::Producer, RowMajor, TileSync<1>>;
  using MiddleCuStage = CuStage<CuStageType::Producer | CuStageType::Consumer, RowMajor, TileSync<1>>;
  using ConsCuStage = CuStage<CuStageType::Consumer, RowMajor, TileSync<1>>;
  using Sync = TileSync<1>;
#elif defined(BATCHEDROW)
  using ProdCuStage = CuStage<CuStageType::Producer, RowMajor, BatchedRowSync>;
  using ConsCuStage = CuStage<CuStageType::Consumer, RowMajor, BatchedRowSync>;
  using Sync = BatchedRowSync;
#else
  #error "Unknown Synchronization"
#endif

#include "common.h"
const uint GLURowTile = 1;

#ifndef EVAL_TILE_SIZES
//Tile sizes of all GeMMs
using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<256, 128, 32>;
using ShapeMMAWarp = cutlass::gemm::GemmShape<128, 64, 32>;
#else
//<eval tiles>
using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<32, 256, 32>;  
using ShapeMMAWarp = cutlass::gemm::GemmShape<32, 64, 32>;
//</eval tiles>
#endif

using ShapeMMAOp = cutlass::gemm::GemmShape<8, 8, 4>;  

//Element types of A, B, and C
using ElementAccumulator = float;
using ElementInputA = cutlass::half_t;
using ElementInputB = cutlass::half_t;
using ElementOutput = cutlass::half_t;
using ElementComputeEpilogue = cutlass::half_t;

//All matrices are in RowMajor
using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::RowMajor;
using LayoutOutput = cutlass::layout::RowMajor;

//Use FP-16 Tensor Cores
using MMAOp = cutlass::arch::OpClassTensorOp;

using SmArch = cutlass::arch::Sm70;

//First GeMM in MLP is fused with GELU
#ifdef LLAMA
using EpilogueOp1 = cutlass::epilogue::thread::LinearCombination<
#elif defined(GPT3)
using EpilogueOp1 = cutlass::epilogue::thread::LinearCombinationGELU<
#endif
    ElementOutput,                                        
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator, 
    ElementComputeEpilogue,                              
    cutlass::epilogue::thread::ScaleType::NoBetaScaling>;

//Second GeMM in MLP performs no extra fused computations 
using EpilogueOp2 = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                        
    128 / cutlass::sizeof_bits<ElementOutput>::value,     
    ElementAccumulator,
    ElementComputeEpilogue>;

template<typename EpilogueOp, bool splitK>
class BaseMLPGemm : public cutlass::gemm::device::Gemm<ElementInputA, LayoutInputA, 
                                                       ElementInputB, LayoutInputB,
                                                       ElementOutput, LayoutOutput,
                                                        ElementAccumulator, MMAOp,
                                                        SmArch, ShapeMMAThreadBlock,
                                                        ShapeMMAWarp, ShapeMMAOp,
                                                        EpilogueOp, 
                                                        cutlass::gemm::threadblock::GemmHorizontalThreadblockSwizzle, 
                                                        2, 8, 8, splitK> {};
// Baseline GeMMs
using Gemm1 = BaseMLPGemm<EpilogueOp1, false>;
using Gemm2 = BaseMLPGemm<EpilogueOp2, false>;

//Baseline GeMMs with SplitK enabled
using GemmSplitK1 = BaseMLPGemm<EpilogueOp1, true>;
using GemmSplitK2 = BaseMLPGemm<EpilogueOp2, true>;

//CuSync GeMMs
using CuSyncImpl = CuSync<ProdCuStage, ConsCuStage>;

using CuSyncImpl1 = CuSync<ProdCuStage, MiddleCuStage>;
using CuSyncImpl2 = CuSync<MiddleCuStage, ConsCuStage>;

template<typename CuStage, typename EpilogueOp, bool splitK>
class CuSyncMLPGemm : public cutlass::gemm::device::CuSyncGemm<CuStage, ElementInputA, LayoutInputA, 
                                                       ElementInputB, LayoutInputB,
                                                       ElementOutput, LayoutOutput,
                                                        ElementAccumulator, MMAOp,
                                                        SmArch, ShapeMMAThreadBlock,
                                                        ShapeMMAWarp, ShapeMMAOp,
                                                        EpilogueOp, 
                                                        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 
                                                        2, 8, 8, splitK> {};

using CuSyncGemm1 = CuSyncMLPGemm<ProdCuStage, EpilogueOp1, false>;
using CuSyncGemm2 = CuSyncMLPGemm<ConsCuStage, EpilogueOp2, false>;

using CuSyncGemmSplitK1 = CuSyncMLPGemm<ProdCuStage, EpilogueOp1, true>;
using CuSyncGemmSplitK2 = CuSyncMLPGemm<ConsCuStage, EpilogueOp2, true>;

using HostTensor = cutlass::HostTensor<ElementInputA, LayoutInputA>;

enum MLPType {
  GPT3,
  LLaMa    
};

struct MLPParameters {
  HostTensor x; //[B, H]
  HostTensor w1; //[H, 4H/8] in GPT-3
  //xw1 = GeLU(x * w1)
  HostTensor xw1; //[B, 4 H / 8]
  HostTensor w2; //[4H/8, H] in GPT-3 and [H/3, H] in LLaMa
  //xw12 = xw1 * w2
  HostTensor xw12; //[B, H]

  //For LLaMa only
  HostTensor vw1; //[B, 2*H/3] in LLAMA
  HostTensor xvw1; //[B, 2*H/3] in LLaMa
  HostTensor glu; //[B, H/3] in LLaMa

  HostTensor ref_xw1;
  HostTensor ref_xw12;

  //For LLaMa only
  HostTensor ref_xv;

  bool checkResults;

  cutlass::gemm::GemmCoord gemm_size1;
  cutlass::gemm::GemmCoord gemm_size2;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;

  std::string model;

  MLPParameters(std::string model_, uint batch, bool check) {
    alpha = ElementComputeEpilogue(1.0);
    beta = ElementComputeEpilogue(0.0);
    model = model_;

    if (model == "gpt3") {
      gemm_size1 = cutlass::gemm::GemmCoord(batch, 4*12288/8, 12288);
      gemm_size2 = cutlass::gemm::GemmCoord(batch, 12288, 4*12288/8);
    } else if (model=="llama") {
      int d = ((8192/3 + 127)/128)*128;
      gemm_size1 = cutlass::gemm::GemmCoord(batch, 2*d, 8192);
      gemm_size2 = cutlass::gemm::GemmCoord(batch, 8192, d);
    }
    std::cout << "GeMM 1 Size: " << gemm_size1.m() << ", " << 
      gemm_size1.n() << ", " << gemm_size1.k() << std::endl;
    std::cout << "GeMM 2 Size: " << gemm_size2.m() << ", " << 
      gemm_size2.n() << ", " << gemm_size2.k() << std::endl;
    
    x = HostTensor(gemm_size1.mk());
    w1 = HostTensor(gemm_size1.kn());
    xw1 = HostTensor(gemm_size1.mn());
    w2 = HostTensor(gemm_size2.kn());
    xw12 = HostTensor(gemm_size2.mn());
    ref_xw1 = HostTensor(gemm_size1.mn());
    ref_xw12 = HostTensor(gemm_size2.mn());

    if (model == "llama") {
      xvw1 = HostTensor(gemm_size1.mn());
      vw1 = HostTensor(gemm_size1.kn());
      glu = HostTensor(gemm_size2.mk());
      ref_xv = HostTensor(gemm_size1.mn());
    }
    checkResults = check;
  }

  void initIns() {
    if (checkResults) {
      ElementOutput values[5] = {ElementOutput(0.05), ElementOutput(0.2), ElementOutput(0.01), ElementOutput(3), ElementOutput(0.4)};
      memset_random(x.host_data(), 5, values, x.size());
      memset_random(w1.host_data(), 5, values, w1.size());
      memset_random2(w2.host_data(), ElementOutput(0.01), ElementOutput(0.05), w2.size());
      if (model == "llama") {
        memset_random2(vw1.host_data(), ElementOutput(0.01), ElementOutput(0.2), vw1.size());
      }
    } else {
      cutlass::reference::host::TensorFill(x.host_view(), ElementOutput(0.05));
      cutlass::reference::host::TensorFill(w1.host_view(), ElementOutput(0.5));
      cutlass::reference::host::TensorFill(w2.host_view(), ElementOutput(0.01));
      if (model == "llama") {
        cutlass::reference::host::TensorFill(vw1.host_view(), ElementOutput(0.5));
      }
    }
    // Copy data from host to GPU
    x.sync_device();
    w1.sync_device();
    w2.sync_device();
    if (model == "llama") {
      vw1.sync_device();
    }
  }
  
  void initOuts() {
    cutlass::reference::host::TensorFill(xw1.host_view());
    cutlass::reference::host::TensorFill(xw12.host_view());
      
    xw1.sync_device();
    xw12.sync_device();
    if (model == "llama") {
      cutlass::reference::host::TensorFill(xvw1.host_view());
      xvw1.sync_device();
      cutlass::reference::host::TensorFill(glu.host_view());
      glu.sync_device();
    }
  }

  void initRefs() {
    cutlass::reference::host::TensorFill(ref_xw12.host_view());
    cutlass::reference::host::TensorFill(ref_xw1.host_view());

    ref_xw12.sync_device();
    ref_xw1.sync_device();
    if (model == "llama") {
      cutlass::reference::host::TensorFill(ref_xv.host_view());
      ref_xv.sync_device(); 
    }
  }

  bool isGPT3() {return model == "gpt3";}
  bool isLLaMa() {return model == "llama";}
};

/** Reference MLP for correctness check **/
hipError_t referenceMLP(MLPParameters& mlpParams) {
  ref_matmul<ElementOutput, ElementAccumulator>(mlpParams.gemm_size1.m(), 
                                                mlpParams.gemm_size1.n(), 
                                                mlpParams.gemm_size1.k(),
                                                mlpParams.x.device_data(), 
                                                mlpParams.w1.device_data(), 
                                                mlpParams.ref_xw1.host_data());
  CUDA_CHECK(hipMemcpy(mlpParams.ref_xw1.device_data(), mlpParams.ref_xw1.host_data(), 
             sizeof(ElementOutput) * mlpParams.ref_xw1.size(), hipMemcpyHostToDevice));
  
  if (mlpParams.isLLaMa()) {
    printf("check not supported in llama\n");
    ref_matmul<ElementOutput, ElementAccumulator>(mlpParams.gemm_size1.m(), 
                                                  mlpParams.gemm_size1.n(), 
                                                  mlpParams.gemm_size1.k(),
                                                  mlpParams.x.device_data(), 
                                                  mlpParams.vw1.device_data(), 
                                                  mlpParams.ref_xv.host_data());
    //Compute XW1 (dot) XV
    for (int b = 0; b < mlpParams.gemm_size1.m(); b++) {
      for (int n = 0; n < mlpParams.gemm_size1.n(); n++) {
        uint index = b * mlpParams.gemm_size1.n() + n;
        mlpParams.ref_xv.host_data()[index] = mlpParams.ref_xw1.host_data()[index] * 
                                              mlpParams.ref_xv.host_data()[index];
      }
    }

    mlpParams.ref_xv.sync_device();

    ref_matmul<ElementOutput, ElementAccumulator>(mlpParams.gemm_size2.m(),
                                                  mlpParams.gemm_size2.n(),
                                                  mlpParams.gemm_size2.k(), 
                                                  mlpParams.ref_xv.device_data(),
                                                  mlpParams.w2.device_data(), 
                                                  mlpParams.ref_xw12.host_data());
  } else {
    ref_matmul<ElementOutput, ElementAccumulator>(mlpParams.gemm_size2.m(),
                                                  mlpParams.gemm_size2.n(),
                                                  mlpParams.gemm_size2.k(), 
                                                  mlpParams.ref_xw1.device_data(),
                                                  mlpParams.w2.device_data(), 
                                                  mlpParams.ref_xw12.host_data());
  }

  return hipSuccess;
}

hipError_t checkMLPResults(MLPParameters& mlpParams) {
  ElementOutput* hostC = new ElementOutput[mlpParams.ref_xw1.size()];
  CUDA_CHECK(hipMemcpy(hostC, mlpParams.xw1.device_data(), 
                        mlpParams.xw1.size() * sizeof(ElementOutput), 
                        hipMemcpyDeviceToHost));
  printf("Checking first GeMM\n");
  bool eq = equals(mlpParams.ref_xw1.size(), mlpParams.ref_xw1.host_data(), hostC, 1e-5);
  if (eq == false) {
    printf("First GeMM not correct\n");
    return hipErrorUnknown;
  }
  printf("First GeMM passed\n");
  ElementOutput* hostE = new ElementOutput[mlpParams.ref_xw12.size()];
  CUDA_CHECK(hipMemcpy(hostE, mlpParams.xw12.device_data(), 
                        mlpParams.xw12.size() * sizeof(ElementOutput), 
                        hipMemcpyDeviceToHost));
  //For LLaMa not checking XV
  printf("Checking second GeMM\n");
  eq = equals(mlpParams.ref_xw12.size(), mlpParams.ref_xw12.host_data(), hostE, 1e-1f);
  if (eq == false) {
    printf("Second GeMM not correct \n");
    return hipErrorUnknown;
  }

  printf("Second GeMM passed\n");

  return hipSuccess;
}

/*GPT3 Baseline MLP*/
template<typename GemmTy1, typename GemmTy2>
hipError_t runBaselineGPT3(int split_k1, int split_k2, 
                            MLPParameters& mlpParams,
                            hipStream_t stream,
                            double& execTime, double& matmul1Time, double& softmaxTime, double& matmul2Time,
                            int iters = 100) {
  //Setup first GeMM
  typename GemmTy1::Arguments args1 {
    mlpParams.gemm_size1,
    mlpParams.x.device_ref(), 
    mlpParams.w1.device_ref(),
    mlpParams.xw1.device_ref(),
    mlpParams.xw1.device_ref(),
    {mlpParams.alpha, mlpParams.beta},
    split_k1};

  size_t workspace_size = GemmTy1::get_workspace_size(args1);
  cutlass::device_memory::allocation<uint8_t> workspace1(workspace_size);
  GemmTy1 gemm_op1;
  cutlass::Status status = gemm_op1.can_implement(args1);
  CUTLASS_CHECK(status);
  status = gemm_op1.initialize(args1, workspace1.get());
  CUTLASS_CHECK(status);

  //Setup Second GeMM
  typename GemmTy2::Arguments args2{ 
    mlpParams.gemm_size2, 
    mlpParams.xw1.device_ref(), 
    mlpParams.w2.device_ref(), 
    mlpParams.xw12.device_ref(), 
    mlpParams.xw12.device_ref(), 
    {mlpParams.alpha, mlpParams.beta},         
    split_k2};
  
  GemmTy2 gemm_op2;
  workspace_size = GemmTy2::get_workspace_size(args2);
  cutlass::device_memory::allocation<uint8_t> workspace2(workspace_size);
  status = gemm_op2.can_implement(args2);
  CUTLASS_CHECK(status);
  status = gemm_op2.initialize(args2, workspace2.get());
  CUTLASS_CHECK(status);
  
  execTime = 0;
  
  //Run kernels
  for (int r = 0; r < iters; r++) {    
    double start = timeInMicroSeconds();
    status = gemm_op1(args1, workspace1.get(), stream);
    CUTLASS_CHECK(status);
    CUDA_CHECK(hipStreamSynchronize(stream));
    double middle1 = timeInMicroSeconds();
    double iterMatMul1 = middle1-start;
    matmul1Time += iterMatMul1;
    status = gemm_op2(args2, workspace2.get(), stream);
    CUTLASS_CHECK(status);
    CUDA_CHECK(hipDeviceSynchronize());
    double middle3 = timeInMicroSeconds();
    double iterMatmul2 = middle3-middle1;
    matmul2Time += iterMatmul2;
    double end = timeInMicroSeconds();
    if (iters > 10)
      printf("{\"Total\": %lf, \"matmul1Time\": %lf, \"matmul2Time\": %lf}\n",end-start,iterMatMul1, iterMatmul2);
    execTime += end-start;
  }

  return hipSuccess;
}

hipError_t runBaselineGPT3(int split_k1, int split_k2, 
                        MLPParameters& mlpParams,
                        hipStream_t stream,
                        double& execTime,
                        double& matmul1Time,
                        double& softmaxTime,
                        double& matmul2Time,
                        int iters = 100) {
  hipError_t result;
  execTime = 0;
  matmul1Time = 0;
  softmaxTime = 0;
  matmul2Time = 0;
  if (split_k1 == 1 && split_k2 == 1) {
    result = runBaselineGPT3<Gemm1, Gemm2>(split_k1, split_k2, mlpParams, stream, execTime, matmul1Time, softmaxTime, matmul2Time, iters);
  } else if (split_k1 > 1 && split_k2 == 1) {
    result = runBaselineGPT3<GemmSplitK1, Gemm2>(split_k1, split_k2, mlpParams, stream, execTime, matmul1Time, softmaxTime, matmul2Time, iters);
  } else if (split_k1 == 1 && split_k2 > 1) {
    result = runBaselineGPT3<Gemm1, GemmSplitK2>(split_k1, split_k2, mlpParams, stream, execTime, matmul1Time, softmaxTime, matmul2Time, iters);
  } else {
    result = runBaselineGPT3<GemmSplitK1, GemmSplitK2>(split_k1, split_k2, mlpParams, stream, execTime, matmul1Time, softmaxTime, matmul2Time, iters);
  }

  return result;
}

/*LLaMA Baseline MLP*/
template<typename T, uint H3>
__global__ void gluKernel(T* xvw1, T* glu) {
  int ROW = blockIdx.x;

  for (int i = threadIdx.x; i < H3; i += blockDim.x) {
    float xw1 = xvw1[ROW * (2 * H3) + i];
    float xv =  xvw1[ROW * (2 * H3) + i + H3];
    glu[ROW * H3 + i] = xw1 * xv;
  }
}

template<typename GemmTy1, typename GemmTy2>
hipError_t runBaselineLLaMA(int split_k1, int split_k2, 
                             MLPParameters& mlpParams,
                             hipStream_t stream1,
                             hipStream_t stream2,
                             double& execTime, double& matmul1Time, 
                             double& matmul2Time, double& matmul3Time,
                             int iters = 100) {
  //Setup XW1 GeMM
  typename GemmTy1::Arguments argsXW1{
    mlpParams.gemm_size1,
    mlpParams.x.device_ref(), 
    mlpParams.w1.device_ref(),
    mlpParams.xvw1.device_ref(),
    mlpParams.xvw1.device_ref(),
    {mlpParams.alpha, mlpParams.beta},
    split_k1};

  size_t workspace_size = GemmTy1::get_workspace_size(argsXW1);
  cutlass::device_memory::allocation<uint8_t> workspace1(workspace_size);
  GemmTy1 gemm_opXVW1;
  cutlass::Status status = gemm_opXVW1.can_implement(argsXW1);
  CUTLASS_CHECK(status);
  status = gemm_opXVW1.initialize(argsXW1, workspace1.get());
  CUTLASS_CHECK(status);

  //Setup XW12 GeMM
  typename GemmTy2::Arguments argsXW12{
    mlpParams.gemm_size2, 
    mlpParams.glu.device_ref(), 
    mlpParams.w2.device_ref(), 
    mlpParams.xw12.device_ref(), 
    mlpParams.xw12.device_ref(), 
    {mlpParams.alpha, mlpParams.beta},         
    split_k2};
  
  GemmTy2 gemm_opXW12;
  workspace_size = GemmTy2::get_workspace_size(argsXW12);
  cutlass::device_memory::allocation<uint8_t> workspace3(workspace_size);
  status = gemm_opXW12.can_implement(argsXW12);
  CUTLASS_CHECK(status);
  status = gemm_opXW12.initialize(argsXW12, workspace3.get());
  CUTLASS_CHECK(status);
  
  execTime = 0; 

  //Run kernels
  for (int r = 0; r < iters; r++) {    
    double start = timeInMicroSeconds();
    status = gemm_opXVW1(stream1);
    CUTLASS_CHECK(status);
    CUDA_CHECK(hipDeviceSynchronize());
    double middle1 = timeInMicroSeconds();
    double iterMatMul1 = middle1-start;
    matmul1Time += iterMatMul1;

    //glu
    gluKernel<half, ((8192/3+127)/128)*128><<<mlpParams.gemm_size1.m(), 
                                              ShapeMMAThreadBlock::kN, 0, stream1>>>
      ((half*)mlpParams.xvw1.device_data(), (half*)mlpParams.glu.device_data());
    CUDA_CHECK(hipDeviceSynchronize());
    double middle2 = timeInMicroSeconds();
    double iterMatMul2 = middle2-middle1;
    matmul2Time += iterMatMul2;

    status = gemm_opXW12(stream1);
    CUTLASS_CHECK(status);
    CUDA_CHECK(hipDeviceSynchronize());
    double middle3 = timeInMicroSeconds();
    double iterMatmul3 = middle3-middle2;
    matmul3Time += iterMatmul3;
    double end = timeInMicroSeconds();
    if (iters > 10)
      printf("{\"Total\": %lf, \"matmul1Time\": %lf, \"matmul2Time\": %lf, \"matmul3Time\": %lf}\n",end-start, iterMatMul1, iterMatMul2, iterMatmul3);
    execTime += end-start;
  }

  return hipSuccess;
}

hipError_t runBaselineLLaMA(int split_k1, int split_k2, 
                        MLPParameters& mlpParams,
                        hipStream_t stream1,
                        hipStream_t stream2,
                        double& execTime,
                        double& matmul1Time,
                        double& matmul2Time,
                        double& matmul3Time,
                        int iters = 100) {
  hipError_t result;
  execTime = 0;
  matmul1Time = 0;
  matmul2Time = 0;
  matmul3Time = 0;
  if (split_k1 == 1 && split_k2 == 1) {
    result = runBaselineLLaMA<Gemm1, Gemm2>(split_k1, split_k2, mlpParams, stream1, stream2, execTime, matmul1Time, matmul2Time, matmul3Time, iters);
  } else if (split_k1 > 1 && split_k2 == 1) {
    result = runBaselineLLaMA<GemmSplitK1, Gemm2>(split_k1, split_k2, mlpParams, stream1, stream2, execTime, matmul1Time, matmul2Time, matmul3Time, iters);
  } else if (split_k1 == 1 && split_k2 > 1) {
    result = runBaselineLLaMA<Gemm1, GemmSplitK2>(split_k1, split_k2, mlpParams, stream1, stream2, execTime, matmul1Time, matmul2Time, matmul3Time, iters);
  } else {
    result = runBaselineLLaMA<GemmSplitK1, GemmSplitK2>(split_k1, split_k2, mlpParams, stream1, stream2, execTime, matmul1Time, matmul2Time, matmul3Time, iters);
  }

  return result;
}


/*CuSync GPT-3 MLP*/
template<typename GemmTy1, typename GemmTy2>
hipError_t runCuSyncGPT3(int split_k1, int split_k2,
                      MLPParameters& mlpParams,
                      CuSyncImpl& handle,
                      hipStream_t producer_stream, 
                      hipStream_t consumer_stream,
                      double& execTime,
                      int iters = 100) {
  typename GemmTy1::Arguments args1{handle.prod(),
                                     mlpParams.gemm_size1,
                                     mlpParams.x.device_ref(),
                                     mlpParams.w1.device_ref(),
                                     mlpParams.xw1.device_ref(),
                                     mlpParams.xw1.device_ref(),
                                     {mlpParams.alpha, mlpParams.beta},         
                                     split_k1};
  GemmTy1 gemm_op1;
  size_t workspace_size = GemmTy1::get_workspace_size(args1);
  cutlass::device_memory::allocation<uint8_t> workspace1(workspace_size);
  cutlass::Status status = gemm_op1.can_implement(args1);
  CUTLASS_CHECK(status);
  status = gemm_op1.initialize(args1, workspace1.get());
  CUTLASS_CHECK(status);

  typename GemmTy2::Arguments args2{handle.cons(),
                                    mlpParams.gemm_size2,  
                                    mlpParams.xw1.device_ref(),
                                    mlpParams.w2.device_ref(),
                                    mlpParams.xw12.device_ref(),
                                    mlpParams.xw12.device_ref(),
                                    {mlpParams.alpha, mlpParams.beta},
                                    split_k2};

  GemmTy2 gemm_op2;
  workspace_size = GemmTy2::get_workspace_size(args2);
  cutlass::device_memory::allocation<uint8_t> workspace2(workspace_size);
  status = gemm_op2.can_implement(args2);
  CUTLASS_CHECK(status);
  status = gemm_op2.initialize(args2, workspace2.get());
  CUTLASS_CHECK(status);

  execTime = 0;
  
  for (int r = 0; r < iters; r++) {
    handle.prod().iter += 1;
    handle.cons().iter += 1;
    gemm_op2.params_.custage.iter += 1;
    gemm_op1.params_.custage.iter += 1;
    
    double start = timeInMicroSeconds();
    status = gemm_op1.run(true, NULL, producer_stream);
    CUTLASS_CHECK(status);

    // CUDA_CHECK(hipDeviceSynchronize());
  #ifndef AVOID_WAIT_KERNEL
    handle.invokeWaitKernel(consumer_stream);
  #endif  
    status = gemm_op2.run(true, NULL, consumer_stream);
    CUTLASS_CHECK(status);
    CUDA_CHECK(hipDeviceSynchronize());
    double end = timeInMicroSeconds();
    if (iters > 10)
      printf("{\"Total\": %lf}\n",end-start);
    execTime += end-start;
  }

  return hipSuccess;
}

hipError_t runCuSyncGPT3(int split_k1, int split_k2, MLPParameters& mlpParams,
                      CuSyncImpl& handle,
                      hipStream_t producer_stream, hipStream_t consumer_stream,
                      double& execTime, int iters = 100) {
  hipError_t result;
  execTime = 0;

  if (split_k1 == 1 && split_k2 == 1) {
    result = runCuSyncGPT3<CuSyncGemm1, CuSyncGemm2>(split_k1, split_k2, mlpParams, handle, producer_stream, consumer_stream, execTime, iters);
  } else if (split_k1 > 1 && split_k2 == 1) {
    result = runCuSyncGPT3<CuSyncGemmSplitK1, CuSyncGemm2>(split_k1, split_k2, mlpParams, handle, producer_stream, consumer_stream, execTime, iters);
  } else if (split_k1 == 1 && split_k2 > 1) {
    result = runCuSyncGPT3<CuSyncGemm1, CuSyncGemmSplitK2>(split_k1, split_k2, mlpParams, handle, producer_stream, consumer_stream, execTime, iters);
  } else {
    result = runCuSyncGPT3<CuSyncGemmSplitK1, CuSyncGemmSplitK2>(split_k1, split_k2, mlpParams, handle, producer_stream, consumer_stream, execTime, iters);
  }

  return result;
}

/**CuSync LLaMa in MLP*/
template<typename T, uint RowTile, uint H3>
__global__ void cusyncgluKernel(uint M, T* xvw1, T* glu, MiddleCuStage cons1, MiddleCuStage prod2) {
  uint ROW = blockIdx.x * RowTile;
  prod2.tile(nullptr);
  for (uint ti = 0; ti < RowTile && ROW < M; ti++) {
    for (uint i = threadIdx.x; i < H3; i += blockDim.x) {
      if (ti == 0) {
        dim3 tile = {ROW/ShapeMMAThreadBlock::kM, i/ShapeMMAThreadBlock::kN, 0};
        cons1.wait(tile);
      }
      float xw1 = xvw1[ROW * (2 * H3) + i];
      float xv =  xvw1[ROW * (2 * H3) + i + H3];
      glu[ROW * H3 + i] = xw1 * xv;
      if (ti == RowTile - 1) {
        dim3 tile = {ROW/ShapeMMAThreadBlock::kM, i/ShapeMMAThreadBlock::kN, 0};
        prod2.post(tile);
      }
    }
    ROW++;
  }
}

template<typename GemmTy1, typename GemmTy2>
hipError_t runCuSyncLLaMA(int split_k1, int split_k2,
                           MLPParameters& mlpParams,
                           CuSyncImpl1& handle1,
                           CuSyncImpl2& handle2,
                           hipStream_t* streams,
                           double& execTime,
                           int iters = 100) {
  typename GemmTy1::Arguments argsXW1{handle1.prod(),
                                      mlpParams.gemm_size1,
                                      mlpParams.x.device_ref(),
                                      mlpParams.w1.device_ref(),
                                      mlpParams.xvw1.device_ref(),
                                      mlpParams.xvw1.device_ref(),
                                      {mlpParams.alpha, mlpParams.beta},         
                                      split_k1};
  GemmTy1 gemm_opXVW1;
  size_t workspace_size = GemmTy1::get_workspace_size(argsXW1);
  cutlass::device_memory::allocation<uint8_t> workspace1(workspace_size);
  cutlass::Status status = gemm_opXVW1.can_implement(argsXW1);
  CUTLASS_CHECK(status);
  status = gemm_opXVW1.initialize(argsXW1, workspace1.get());
  CUTLASS_CHECK(status);

  typename GemmTy2::Arguments argsXW12{handle2.cons(),
                                       mlpParams.gemm_size2,  
                                       mlpParams.xvw1.device_ref(),
                                       mlpParams.w2.device_ref(),
                                       mlpParams.xw12.device_ref(),
                                       mlpParams.xw12.device_ref(),
                                       {mlpParams.alpha, mlpParams.beta},
                                       split_k2};

  GemmTy2 gemm_opXW12;
  workspace_size = GemmTy2::get_workspace_size(argsXW12);
  cutlass::device_memory::allocation<uint8_t> workspace3(workspace_size);
  status = gemm_opXW12.can_implement(argsXW12);
  CUTLASS_CHECK(status);
  status = gemm_opXW12.initialize(argsXW12, workspace3.get());
  CUTLASS_CHECK(status);

  execTime = 0;
  
  for (int r = 0; r < iters; r++) {
    handle1.prod().iter += 1;
    handle1.cons().iter += 1;
    handle2.prod().iter += 1;
    handle2.cons().iter += 1;

    gemm_opXW12.params_.custage.iter += 1;
    gemm_opXVW1.params_.custage.iter += 1;

    double start = timeInMicroSeconds();
    status = gemm_opXVW1.run(true, NULL, streams[0]);
    CUTLASS_CHECK(status);

  #ifndef AVOID_WAIT_KERNEL
    handle1.invokeWaitKernel(streams[1]);
  #endif
    //glu
    cusyncgluKernel<half, GLURowTile, ((8192/3+127)/128)*128>
      <<<mlpParams.gemm_size1.m(), ShapeMMAThreadBlock::kN, 0, streams[1]>>>
      (mlpParams.gemm_size1.m(), (half*)mlpParams.xvw1.device_data(), 
       (half*)mlpParams.glu.device_data(), handle1.cons(), handle2.prod());

  #ifndef AVOID_WAIT_KERNEL
    handle2.invokeWaitKernel(streams[2]);
  #endif  
    status = gemm_opXW12.run(true, NULL, streams[2]);
    CUTLASS_CHECK(status);
    CUDA_CHECK(hipDeviceSynchronize());

    double end = timeInMicroSeconds();
    if (iters > 10)
      printf("{\"Total\": %lf}\n",end-start);
    execTime += end-start;
  }

  return hipSuccess;
}

hipError_t runCuSyncLLaMA(int split_k1, int split_k2, 
                          MLPParameters& mlpParams,
                          CuSyncImpl1& handle1,
                          CuSyncImpl2& handle2,
                          hipStream_t* streams,
                          double& execTime, int iters = 100) {
  hipError_t result;
  execTime = 0;

  if (split_k1 == 1 && split_k2 == 1) {
    result = runCuSyncLLaMA<CuSyncGemm1, CuSyncGemm2>(split_k1, split_k2, mlpParams, handle1, handle2, streams, execTime, iters);
  } else if (split_k1 > 1 && split_k2 == 1) {
    result = runCuSyncLLaMA<CuSyncGemmSplitK1, CuSyncGemm2>(split_k1, split_k2, mlpParams, handle1, handle2, streams, execTime, iters);
  } else if (split_k1 == 1 && split_k2 > 1) {
    result = runCuSyncLLaMA<CuSyncGemm1, CuSyncGemmSplitK2>(split_k1, split_k2, mlpParams, handle1, handle2, streams, execTime, iters);
  } else {
    result = runCuSyncLLaMA<CuSyncGemmSplitK1, CuSyncGemmSplitK2>(split_k1, split_k2, mlpParams, handle1, handle2, streams, execTime, iters);
  }

  return result;
}

int run(int argc, char* argv[]) {
  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major != 7) {
    std::cerr << "Volta Tensor Ops must be run on a machine"
              << "with compute capability of 70, 72, or 75."
              << std::endl;
    return 0;
  }
  
  const uint NUM_ARGS = 5;
  std::string argNames[NUM_ARGS] = {"--model", "--batch", "--check", "--split-k1", "--split-k2"};
  std::string argHelp[NUM_ARGS] = {"GPT3 or LLaMa", "Batch size", "Check results", 
                                   "Split K for first GeMM", "Split K for second GeMM"};
  
  if (argc < NUM_ARGS+1) {
    std::cout << "usage: " << std::endl
              << argNames[0] << " gpt3|llama " << argHelp[0] << std::endl 
              << argNames[1] << " <int>" << argHelp[1] << std::endl
              << argNames[2] << " true|false" << argHelp[2] << std::endl
              << argNames[3] << " <int> " << argHelp[3] << std::endl
              << argNames[4] << " <int> " << argHelp[4] << std::endl;
    return 0;
  }

  std::string model = "";
  uint batch = 0;
  bool doChecking = false;
  uint split_k1 = 1;
  uint split_k2 = 1;

  for (int i = 1; i < argc; ++i) {
    std::string arg = std::string(argv[i]);
    if (arg.find(argNames[0]) == 0) {
      model = std::string(argv[i+1]);
      i = i + 1;
    } else if (arg.find(argNames[1]) == 0) {
      std::stringstream ss(argv[i+1]);
      ss >> batch;
      i = i + 1;
    } else if (arg.find(argNames[2]) == 0) {
      std::string val = std::string(argv[i+1]);
      if (val == "true") {
        doChecking = true;
      } else if (val == "false") {
        doChecking = false;
      } else {
        std::cout << "Invalid value for check " << val << std::endl;
      }
      i = i + 1;
    } else if (arg.find(argNames[3]) == 0) {
      split_k1 = atoi(argv[i+1]);
      i=i+1;
    } else if (arg.find(argNames[4]) == 0) {
      split_k2 = atoi(argv[i+1]);
      i=i+1;
    }
  }

  if (model == "" || batch == 0) {
    std::cout<<"invalid model or batch" <<std::endl;
    return 0;
  }
    
  std::cout << "model=" << model << " batch=" << batch << " check="<<doChecking <<std::endl;

  hipStream_t producer_stream;
  hipStream_t producer_stream2;
  hipStream_t consumer_stream;
  CUDA_CHECK(hipStreamCreate(&producer_stream));
  CUDA_CHECK(hipStreamCreate(&producer_stream2));
  CUDA_CHECK(hipStreamCreate(&consumer_stream));

  MLPParameters mlpParams(model, batch, doChecking);
  mlpParams.initIns();
  mlpParams.initOuts();
  mlpParams.initRefs();
  
  hipError_t result;
  int epochs = 20;
  int warmup = 10;

  if (doChecking) {
    //Run our reference MLP
    result = referenceMLP(mlpParams);
    if (result != hipSuccess) {
      return 1;
    }
  }

  //Run baseline MLP
  double baselineTime = 0;
  double matmul1Time = 0;
  double softmaxTime = 0;
  double matmul2Time = 0;

  if (mlpParams.isGPT3()) {
    result = runBaselineGPT3(split_k1, split_k2, mlpParams, producer_stream, 
                             baselineTime, matmul1Time, softmaxTime, matmul2Time, 1);

    CUDA_CHECK(hipDeviceSynchronize());

    if (doChecking) {
      result = checkMLPResults(mlpParams);
      if (result != hipSuccess) {
        return 1;
      }
    }

    result = runBaselineGPT3(split_k1, split_k2, mlpParams, producer_stream, 
                             baselineTime, matmul1Time, softmaxTime, matmul2Time, warmup);

    CUDA_CHECK(hipDeviceSynchronize());
    printf("START-BASELINE:\n");
    result = runBaselineGPT3(split_k1, split_k2, mlpParams, producer_stream, 
                         baselineTime, matmul1Time, softmaxTime, matmul2Time, epochs);
    CUDA_CHECK(result);
    printf("END-BASELINE:\n");
    printf("Average time %lf microseconds\n", baselineTime/(float)epochs);
  } else if (mlpParams.isLLaMa()) {
    result = runBaselineLLaMA(split_k1, split_k2, mlpParams, producer_stream, 
                              producer_stream2, baselineTime, matmul1Time, softmaxTime, matmul2Time, 1);

    CUDA_CHECK(hipDeviceSynchronize());

    if (doChecking) {
      result = checkMLPResults(mlpParams);
      if (result != hipSuccess) {
        return 1;
      }
    }

    result = runBaselineLLaMA(split_k1, split_k2, mlpParams, producer_stream, 
                              producer_stream2, baselineTime, matmul1Time, softmaxTime, matmul2Time, warmup);

    CUDA_CHECK(hipDeviceSynchronize());
    printf("START-BASELINE:\n");
    result = runBaselineLLaMA(split_k1, split_k2, mlpParams, producer_stream, 
                              producer_stream2, baselineTime, matmul1Time, softmaxTime, matmul2Time, epochs);
    CUDA_CHECK(result);
    printf("END-BASELINE:\n");
    printf("Average time %lf microseconds\n", baselineTime/(float)epochs);
  }

  
  if (doChecking) {
    mlpParams.initOuts();
  }
  printf("mlpParams.gemm_size1.n() %d ShapeMMAThreadBlock::kN %d\n", mlpParams.gemm_size1.n(), ShapeMMAThreadBlock::kN);
  //Setup cusync gemm
  dim3 gridDim1 = {(uint)DIVUP(mlpParams.gemm_size1.m(), ShapeMMAThreadBlock::kM), 
                  (uint)DIVUP(mlpParams.gemm_size1.n(), ShapeMMAThreadBlock::kN), 
                  split_k1};
  dim3 gridDim2 = {(uint)DIVUP(mlpParams.gemm_size2.m(), ShapeMMAThreadBlock::kM), 
                   (uint)DIVUP(mlpParams.gemm_size2.n(), ShapeMMAThreadBlock::kN), 
                   split_k2};
  dim3 tileSize = {ShapeMMAThreadBlock::kM, ShapeMMAThreadBlock::kN, 1};

#if defined(ROWSYNC)
  using Sync = RowSync;
  uint waitValue = gridDim1.y;
  RowSync sync(gridDim1.y);
#elif defined(TILEBATCH)
  using Sync = TileSync<2>;
  Sync sync;
#elif defined(TILESYNC)
  using Sync = TileSync<1>;
  Sync sync;
#elif defined(BATCHEDROW)
  using Sync = BatchedRowSync;
  BatchedRowSync sync(gridDim1.y, 1);
#else
  #error "Unknown Policy"
#endif

  int highestPriority;
  int lowestPriority;
  CUDA_CHECK(hipDeviceGetStreamPriorityRange(&lowestPriority, &highestPriority));
  CUDA_CHECK(hipStreamCreateWithPriority(&consumer_stream, 0, lowestPriority));
  hipStream_t streams[(lowestPriority - highestPriority + 1)];
  for (int i = highestPriority; i <= lowestPriority; i++) {
    CUDA_CHECK(hipStreamCreateWithPriority(&streams[i - highestPriority], 0, i));
  }
  
  //Run cusync mlp
  if (mlpParams.isGPT3()) {
    ProdCuStage prod(gridDim1, tileSize, sync);
    ConsCuStage cons(gridDim2, tileSize, sync);

    prod.iter = cons.iter = 0;

    CuSyncImpl cuSyncHandle(prod, cons);
    
    double overlapTime = 0;
    cuSyncHandle.iter = 0;
    cuSyncHandle.prod().iter = cuSyncHandle.cons().iter = 0;

    result = runCuSyncGPT3(split_k1, split_k2, mlpParams, cuSyncHandle, producer_stream, consumer_stream, overlapTime, 1);

    CUDA_CHECK(hipDeviceSynchronize());
    if (doChecking) {
      result = checkMLPResults(mlpParams);
      if (result != hipSuccess) {
        return 1;
      }
    }

    result = runCuSyncGPT3(split_k1, split_k2, mlpParams, cuSyncHandle, producer_stream, consumer_stream, overlapTime, warmup);
    
    CUDA_CHECK(hipDeviceSynchronize());
    printf("START-OVERLAPPED:\n");
    
    result = runCuSyncGPT3(split_k1, split_k2, mlpParams, cuSyncHandle, producer_stream, consumer_stream, overlapTime, epochs);
    
    CUDA_CHECK(result);
    printf("END-OVERLAPPED:\n");
    
    printf("Average time %lf microseconds\n", overlapTime/(float)epochs);
  } else if (mlpParams.isLLaMa()) {
#if defined(ROWSYNC)
  RowSync sync2(min(ShapeMMAThreadBlock::kM, mlpParams.gemm_size1.m()), GLURowTile);
#elif defined(TILESYNC)
  using Sync = TileSync<1>;
  uint waitValue = DIVUP(min(mlpParams.gemm_size1.m(), ShapeMMAThreadBlock::kM), GLURowTile);
  Sync sync2(waitValue, 1);
#else
  #error "Unknown Policy"
#endif
    ProdCuStage prod(gridDim1, tileSize, sync);
    dim3 gridMiddle = {(uint)DIVUP(mlpParams.gemm_size1.m(), GLURowTile), 1, 1};
    MiddleCuStage middle(gridMiddle, {GLURowTile, 1, 1}, sync);
    ConsCuStage cons(gridDim2, tileSize, sync2);

    prod.iter = cons.iter = middle.iter = 0;

    CuSyncImpl1 cuSyncHandle1(prod, middle);
    CuSyncImpl2 cuSyncHandle2(middle, cons);
    
    double overlapTime = 0;
    cuSyncHandle1.iter = 0;
    cuSyncHandle1.prod().iter = cuSyncHandle1.cons().iter = 0;
    cuSyncHandle2.iter = 0;
    cuSyncHandle2.prod().iter = cuSyncHandle2.cons().iter = 0;

    // cuSyncHandle2.prod().setTileStatusToWait(cuSyncHandle1.cons().getTileStatusToWait());
    cuSyncHandle1.cons().kernelExecuted_ = cuSyncHandle2.prod().kernelExecuted_;

    result = runCuSyncLLaMA(split_k1, split_k2, mlpParams, cuSyncHandle1, cuSyncHandle2, streams, overlapTime, 1);

    CUDA_CHECK(hipDeviceSynchronize());
    if (doChecking) {
      result = checkMLPResults(mlpParams);
      if (result != hipSuccess) {
        return 1;
      }
    }

    result = runCuSyncLLaMA(split_k1, split_k2, mlpParams, cuSyncHandle1, cuSyncHandle2, streams, overlapTime, warmup);
    
    CUDA_CHECK(hipDeviceSynchronize());
    printf("START-OVERLAPPED:\n");
    
    result = runCuSyncLLaMA(split_k1, split_k2, mlpParams, cuSyncHandle1, cuSyncHandle2, streams, overlapTime, epochs);
    
    CUDA_CHECK(result);
    printf("END-OVERLAPPED:\n");
    
    printf("Average time %lf microseconds\n", overlapTime/(float)epochs);
  }

  return 0;
}
