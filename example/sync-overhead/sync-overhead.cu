
#include <hip/hip_runtime.h>
#include<stdio.h>

#include <time.h>
#include <sys/time.h>


#define CUDA_CHECK(cmd) do {                        \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0);

static double convertTimeValToDouble(struct timeval _time) {
  return ((double)_time.tv_sec)*1e6 + ((double)_time.tv_usec);
}

static struct timeval getTimeOfDay () {
  struct timeval _time;

  if (gettimeofday (&_time, NULL) == -1) {
    fprintf (stderr, "gettimeofday returned -1\n");
    perror ("");
    abort ();
  }

  return _time;
}

static double timeInMicroSeconds() {
  return convertTimeValToDouble(getTimeOfDay());
}

static double getCurrentTime() {
  return timeInMicroSeconds();
}

__global__ void kernel1(float *in, int i, volatile int* sync) {
	int linearid = threadIdx.x + blockIdx.x * blockDim.x;
	in[linearid] = i;
  __syncthreads();
  if (threadIdx.x == 0)
    sync[blockIdx.x] = 1;
}

__global__ void kernel2(float *out, float *in, volatile int* sync, bool cansync) {
	if (cansync && threadIdx.x == 0) {
		while (sync[blockIdx.x] != 1);
		sync[blockIdx.x] = 0;
	}
	__syncthreads();
	int linearid = threadIdx.x + blockIdx.x * blockDim.x;
	out[linearid] = in[linearid] + 1;
}

int main() {
  float* in, *out;
  size_t size = 1 << 20;
	CUDA_CHECK(hipMalloc(&in, size));
  CUDA_CHECK(hipMalloc(&out, size));
  int* sync;
  CUDA_CHECK(hipMalloc(&sync, size));
  CUDA_CHECK(hipMemset(sync, size * sizeof(int), 0));
  unsigned int threads = 128;
  dim3 grid = {80*2 * (1024/threads), 1, 1};
  dim3 block = {threads, 1, 1}; 
  hipStream_t prodstream, constream;

  int highestPriority;
  int lowestPriority;
  
  CUDA_CHECK(hipDeviceGetStreamPriorityRange(&lowestPriority, &highestPriority));

  CUDA_CHECK(hipStreamCreateWithPriority(&prodstream, 0, highestPriority));
  CUDA_CHECK(hipStreamCreateWithPriority(&constream, 0, lowestPriority));
  CUDA_CHECK(hipDeviceSynchronize());
  double sync_exec = 0;
  for (int i = 0; i < 110; i++) {
    double s = getCurrentTime();
    kernel1<<<grid,block,0,prodstream>>>(in, 0, sync);
    kernel2<<<grid,block,0,prodstream>>>(out, in, sync, true);
    CUDA_CHECK(hipDeviceSynchronize());
    double t = getCurrentTime();
    if (i >= 10)
      sync_exec += t - s;
  }

  printf("exec with sync %lf\n", sync_exec);
  CUDA_CHECK(hipDeviceSynchronize());

  double exec = 0;
  for (int i = 0; i < 100; i++) {
    double s = getCurrentTime();
    kernel1<<<grid,block>>>(in, 0, sync);
    kernel2<<<grid,block>>>(out, in, sync, false);
    CUDA_CHECK(hipDeviceSynchronize());
    double t = getCurrentTime();
    exec += t - s;
  }
  printf("exec without sync %lf\n", exec);
  printf("Overhead %lf\n", (sync_exec - exec)/exec);
}