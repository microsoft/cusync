#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Based on the example from NVIDIA CUDA Samples 
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication which makes use of shared memory
 * to ensure data reuse, the matrix multiplication is done using tiling
 * approach. It has been written for clarity of exposition to illustrate various
 * CUDA programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication. See also: V. Volkov and
 * J. Demmel, "Benchmarking GPUs to tune dense linear algebra," in Proc. 2008
 * ACM/IEEE Conf. on Supercomputing (SC '08), Piscataway, NJ: IEEE Press, 2008,
 * pp. Art. 31:1-11.
 */
 
// System includes
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// CuSync include
#include <cuSync.h>

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
using ProdCuStage = CuStage<CuStageType::Producer, RowMajor, TileSync<1>>;
using ConsCuStage = CuStage<CuStageType::Consumer, RowMajor, TileSync<1>>;

template <typename CuStageTy, int BLOCK_SIZE>
__global__ void MatrixMulCUDA(CuStageTy custage, float *C, float *A,
                              float *B, int wA, int wB) {
  __shared__ int tileSh[3];
  // Get tile to compute by this thread block
  dim3 tile = custage.tile((dim3*)&tileSh[0]);

  // Block index
  int bx = tile.x;
  int by = tile.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    // Wait for tile of A to be computed by producer kernel
    dim3 tile = {(a - aBegin)/BLOCK_SIZE, by, 1};
    custage.wait(tile);

    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;

  // Post the status of tile when computed
  custage.post(tile);
}

void ConstantInit(float *data, int size, float val) {
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv, int block_size, const dim3 &dimsA,
                   const dim3 &dimsB, const dim3 &dimsD) {
  // Allocate host memory for matrices A and B
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float *h_A;
  CUDA_CHECK(hipHostMalloc(&h_A, mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B;
  CUDA_CHECK(hipHostMalloc(&h_B, mem_size_B));
  float *h_D;
  CUDA_CHECK(hipHostMalloc(&h_D, mem_size_A));
  
  hipStream_t prod_stream, cons_stream;

  // Initialize host memory
  const float valB = 0.01f;
  ConstantInit(h_A, size_A, 1.0f);
  ConstantInit(h_B, size_B, valB);
  ConstantInit(h_D, size_B, valB);

  // Allocate device memory
  float *d_A, *d_B, *d_C, *d_D, *d_E;

  // Allocate host matrix C and E
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
  float *h_C;
  CUDA_CHECK(hipHostMalloc(&h_C, mem_size_C));

  dim3 dimsE(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_E = dimsC.x * dimsC.y * sizeof(float);
  float *h_E;
  CUDA_CHECK(hipHostMalloc(&h_E, mem_size_E));

  if (h_C == NULL) {
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    exit(EXIT_FAILURE);
  }

  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D), mem_size_B));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_E), mem_size_E));

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipStreamCreateWithFlags(&cons_stream, hipStreamNonBlocking));
  CUDA_CHECK(hipStreamCreateWithFlags(&prod_stream, hipStreamNonBlocking));

  // copy host memory to device
  CUDA_CHECK(
      hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(d_D, h_D, mem_size_B, hipMemcpyHostToDevice));
  
  // Setup execution parameters
  dim3 threads(block_size, block_size, 1);
  dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y, 1);
  
  // Create CuSync and CuStage
  TileSync<1> sync;
  dim3 tilesize = threads;
  ProdCuStage prod(grid, tilesize, sync);
  ConsCuStage cons(grid, tilesize, sync);
  prod.iter = cons.iter = 1;
  initProducerConsumer(prod, cons);

  // Create and start timer
  printf("Computing result using CUDA Kernel...\n");

  assert (block_size == 32);
  // Invoke producer kernel (C = A * B)
  MatrixMulCUDA<ProdCuStage, 32>
        <<<grid, threads, 0, prod_stream>>>(prod, d_C, d_A, d_B, dimsA.x, dimsB.x);

  //Invoke wait kernel
  prod.invokeWaitKernel(cons_stream);
  // //Invoke consumer kernel (E = C * D)
  // CUDA_CHECK(hipDeviceSynchronize());

  MatrixMulCUDA<ConsCuStage, 32>
        <<<grid, threads, 0, cons_stream>>>(cons, d_E, d_C, d_D, dimsA.x, dimsB.x);
  
  CUDA_CHECK(hipDeviceSynchronize());
  printf("done\n");
  
  // Copy result from device to host
  CUDA_CHECK(
      hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));
  CUDA_CHECK(
      hipMemcpy(h_E, d_E, mem_size_C, hipMemcpyDeviceToHost));

  printf("Checking computed result for correctness: \n");
  bool correct = true;

  // test relative error by the formula
  //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  double eps = 1.e-5;  // machine zero
  // Check C
  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
    double abs_err = fabs(h_C[i] - (dimsA.x * valB));
    double dot_length = dimsA.x;
    double abs_val = fabs(h_C[i]);
    double rel_err = abs_err / abs_val / dot_length;

    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i,
             h_C[i], dimsA.x * valB, eps);
      correct = false;
      break;
    }
  }

  printf("C results: %s\n", correct ? "PASS" : "FAIL");

  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
    double abs_err = fabs(h_E[i] - (dimsA.x * valB * dimsA.x * valB));
    double dot_length = dimsA.x;
    double abs_val = fabs(h_E[i]);
    double rel_err = abs_err / abs_val / dot_length;

    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i,
             h_E[i], dimsA.x * valB  * dimsA.x, eps);
      correct = false;
      break;
    }
  }

  printf("E results: %s\n", correct ? "PASS" : "FAIL");

  // Clean up memory
  CUDA_CHECK(hipHostFree(h_A));
  CUDA_CHECK(hipHostFree(h_B));
  CUDA_CHECK(hipHostFree(h_C));
  CUDA_CHECK(hipHostFree(h_D));
  CUDA_CHECK(hipHostFree(h_E));
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));
  CUDA_CHECK(hipFree(d_D));
  CUDA_CHECK(hipFree(d_E));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  if (correct) {
    return EXIT_SUCCESS;
  } else {
    return EXIT_FAILURE;
  }
}

/**
 * Program main
 */
int main(int argc, char **argv) {
  printf("[Matrix Multiply Using CUDA] - Starting...\n");

  // This will pick the best possible CUDA capable device, otherwise
  // override the device ID based on input provided at the command line

  int block_size = 32;

  dim3 dimsA(4 * 2 * block_size, 4 * 2 * block_size, 1);
  dim3 dimsB = dimsA;
  dim3 dimsD = dimsA;

  printf("MatrixA(%d,%d), MatrixB(%d,%d), MatrixD(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x,
         dimsB.y, dimsD.x, dimsD.y);

  int matrix_result = MatrixMultiply(argc, argv, block_size, dimsA, dimsB, dimsD);

  exit(matrix_result);
}
